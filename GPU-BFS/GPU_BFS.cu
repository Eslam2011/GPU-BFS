#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include<time.h>

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <conio.h>
#define NUM_NODES 6
#define num_blks 1


typedef struct
{
	int start;     // Index of first adjacent neigbour node in d_adjLists	
	int length;    // Number of neighbour nodes 
} Node;

__global__ void CUDA_BFS_KERNEL(Node* d_VertixArray, int* d_adjLists, bool* d_front, bool* d_Visited, bool* done)
{

	int id = threadIdx.x + blockIdx.x * blockDim.x;
	if (id > NUM_NODES)
		*done = false;
	

	if (d_front[id] == true && d_Visited[id] == false)
	{
		 printf("%d ", id); 	
		 d_front[id] = false;
		 d_Visited[id] = true;
		__syncthreads();
		//	int k = 0;
			//int i;
		int start = d_VertixArray[id].start;
		int end = start + d_VertixArray[id].length;
		for (int i = start; i < end; i++)
		{
			int nid =  d_adjLists[i];

			if (d_Visited[nid] == false && d_front[nid] == false && d_Visited[id] == true)
			{
				//printf("%d", nid);
				d_front[nid] = true;
				*done = false;
			}

		}

	}

}


int main()
{
	Node Vertex[NUM_NODES];
	int edges[15];
	hipEvent_t start, stop;
	Node* d_VertexArray;
	int* d_adjLists;
	bool done;
	bool* d_done;
	bool* d_front;
	bool* d_Visited;



	Vertex[0].start = 0;
	Vertex[0].length = 2;

	Vertex[1].start = 2;
	Vertex[1].length = 3;

	Vertex[2].start = 5;
	Vertex[2].length = 3;

	Vertex[3].start = 8;
	Vertex[3].length = 3;

	Vertex[4].start = 11;
	Vertex[4].length = 2;

	Vertex[5].start = 13;
	Vertex[5].length = 2;

	edges[0] = 1;
	edges[1] = 2;
	edges[2] = 0;
	edges[3] = 3;
	edges[4] = 0;
	edges[5] = 0;
	edges[6] = 3;
	edges[7] = 5;
	edges[8] = 1;
	edges[9] = 2;
	edges[10] = 4;
	edges[11] = 3;
	edges[12] = 5;
	edges[13] = 2;
	edges[14] = 4;

	bool front[NUM_NODES] = { false };
	bool visited[NUM_NODES] = { false };


	int source = 0;
	front[source] = true;

	 
	hipMalloc((void**)&d_VertexArray, sizeof(Node) * NUM_NODES);
	hipMemcpy(d_VertexArray, Vertex, sizeof(Node) * NUM_NODES, hipMemcpyHostToDevice);

	hipMalloc((void**)&d_adjLists, sizeof(Node) * NUM_NODES);
	hipMemcpy(d_adjLists, edges, sizeof(Node) * NUM_NODES, hipMemcpyHostToDevice);

	hipMalloc((void**)&d_front, sizeof(bool) * NUM_NODES);
	hipMemcpy(d_front, front, sizeof(bool) * NUM_NODES, hipMemcpyHostToDevice);

	hipMalloc((void**)&d_Visited, sizeof(bool) * NUM_NODES);
	hipMemcpy(d_Visited, visited, sizeof(bool) * NUM_NODES, hipMemcpyHostToDevice);

	hipMalloc((void**)&d_done, sizeof(bool));

	//int count = 0;
	printf("Breadth-First Search: ");
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	do {
		//count++;
		done = true;
		hipMemcpy(d_done, &done, sizeof(bool), hipMemcpyHostToDevice);
		CUDA_BFS_KERNEL << <num_blks, NUM_NODES >> > (d_VertexArray, d_adjLists, d_front, d_Visited, d_done);
		hipMemcpy(&done, d_done, sizeof(bool), hipMemcpyDeviceToHost);
	} while (!done);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("\nGPU Time: %f s", elapsedTime / 1000);
	hipEventDestroy(start);
	hipEventDestroy(stop);


	hipFree(d_done);
	hipFree(d_VertexArray);
	hipFree(d_adjLists);
	hipFree(d_front);
	hipFree(d_Visited);

}